#include "hip/hip_runtime.h"
#include "mat4_col_major.h"
#include "aabb2.h"
//
#include "tile_acceleration.h"

extern "C" {

struct Splat3{
    float xyz[3];
    unsigned char rgb[3];
};

struct Splat2 {
    float z;
    float pos_pix[2];
    float rad;
    float rgb[3];
};

__global__
void splat3_to_splat2(
  uint32_t num_pnt,
  Splat2* pnt2splat,
  const Splat3 *pnt2xyzrgb,
  const float *transform_world2ndc,
  const uint32_t img_w,
  const uint32_t img_h,
  float radius)
{
    int i_pnt = blockDim.x * blockIdx.x + threadIdx.x;
    if( i_pnt >= num_pnt ){ return; }
    //
    const auto p0 = pnt2xyzrgb[i_pnt].xyz;
    const auto q0 = mat4_col_major::transform_homogeneous(
        transform_world2ndc, p0);
   float r0[2] = {
     (q0[0] + 1.f) * 0.5f * float(img_w),
     (1.f - q0[1]) * 0.5f * float(img_h) };
   float rad;
   {
       const cuda::std::array<float,9> dqdp = mat4_col_major::jacobian_transform(transform_world2ndc, p0);
       const cuda::std::array<float,9> dpdq = mat3_col_major::try_inverse(dqdp.data()).value();
       const float dx[3] = { dpdq[0], dpdq[1], dpdq[2] };
       const float dy[3] = { dpdq[3], dpdq[4], dpdq[5] };
       float rad_pix_x = (1.f / vec3::norm(dx)) * 0.5f * float(img_w) * radius;
       float rad_pxi_y = (1.f / vec3::norm(dy)) * 0.5f * float(img_h) * radius;
       rad = 0.5f * (rad_pix_x + rad_pxi_y);
   }
   pnt2splat[i_pnt].z = q0[2];
   pnt2splat[i_pnt].pos_pix[0] = r0[0];
   pnt2splat[i_pnt].pos_pix[1] = r0[1];
   pnt2splat[i_pnt].rad = rad;
   pnt2splat[i_pnt].rgb[0] = float(pnt2xyzrgb[i_pnt].rgb[0]) / 255.0;
   pnt2splat[i_pnt].rgb[1] = float(pnt2xyzrgb[i_pnt].rgb[1]) / 255.0;
   pnt2splat[i_pnt].rgb[2] = float(pnt2xyzrgb[i_pnt].rgb[2]) / 255.0;
}



__global__
void count_splat_in_tile(
  uint32_t num_pnt,
  const Splat2* pnt2splat,
  uint32_t* tile2ind,
  uint32_t* pnt2ind,
  uint32_t tile_w,
  uint32_t tile_h,
  uint32_t tile_size)
{
    int i_pnt = blockDim.x * blockIdx.x + threadIdx.x;
    if( i_pnt >= num_pnt ){ return; }
    //
    const Splat2& splat = pnt2splat[i_pnt];
    const cuda::std::array<float,4> aabb = aabb2::from_point(splat.pos_pix, splat.rad);
    //
    tile_acceleration::count_splat_in_tile(
        i_pnt, aabb,
        tile2ind, pnt2ind,
        tile_w, tile_h, tile_size);
}

__global__
void fill_index_info(
  uint32_t num_pnt,
  const Splat2* pnt2splat,
  const uint32_t* pnt2idx,
  uint64_t* idx2tiledepth,
  uint32_t* idx2pnt,
  uint32_t tile_w,
  uint32_t tile_h,
  uint32_t tile_size)
{
    int i_pnt = blockDim.x * blockIdx.x + threadIdx.x;
    if( i_pnt >= num_pnt ){ return; }
    //
    const Splat2& splat = pnt2splat[i_pnt];
    const cuda::std::array<float,4> aabb = aabb2::from_point(splat.pos_pix, splat.rad);
    tile_acceleration::fill_index_info(
        i_pnt, aabb, splat.z,
        pnt2idx, idx2tiledepth, idx2pnt,
        tile_w, tile_h, tile_size);
}

__global__
void rasterize_splat_using_tile(
    uint32_t img_w,
    uint32_t img_h,
    float* d_pix2rgb,
    uint32_t tile_w,
    uint32_t tile_h,
    uint32_t tile_size,
    const uint32_t* d_tile2idx,
    const uint32_t* d_idx2pnt,
    const Splat2* d_pnt2splat)
{
    const uint32_t ix = blockDim.x * blockIdx.x + threadIdx.x;
    if( ix >= img_w ){ return; }
    //
    const uint32_t iy = blockDim.y * blockIdx.y + threadIdx.y;
    if( iy >= img_h ){ return; }
    const uint32_t i_pix = iy * img_w + ix;
    //
    const uint32_t i_tile = (iy / tile_size) * tile_w + (ix / tile_size);
    for(uint32_t idx=d_tile2idx[i_tile]; idx<d_tile2idx[i_tile+1];++idx){
        const uint32_t i_pnt = d_idx2pnt[idx];
        const Splat2& splat = d_pnt2splat[i_pnt];
        const float p0[2] = {
            float(ix) + 0.5f,
            float(iy) + 0.5f};
        const float dx = splat.pos_pix[0] - p0[0];
        const float dy = splat.pos_pix[1] - p0[1];
        const float distance = sqrt(dx * dx + dy * dy);
        if( distance > splat.rad ){ continue; }
        d_pix2rgb[i_pix*3+0] = splat.rgb[0];
        d_pix2rgb[i_pix*3+1] = splat.rgb[1];
        d_pix2rgb[i_pix*3+2] = splat.rgb[2];
    }

}


} // extern "C"