#include "hip/hip_runtime.h"
#include "mat2_sym.h"
#include "mat4_col_major.h"
#include "mat2x3_col_major.h"
#include "quaternion.h"
#include "aabb2.h"
#include "tile_acceleration.h"

extern "C" {

struct Splat3 {
    float xyz[3];
    float rgb_dc[3];
    float rgb_sh[45];
    float opacity;
    float scale[3];
    float quaternion[4];
};

struct Splat2 {
    float pos_pix[2];
    float sig_inv[3];
    float aabb[4];
    float rgb[3];
    float alpha;
    float ndc_z;
};

__global__
void splat3_to_splat2(
  uint32_t num_pnt,
  Splat2* pnt2splat2,
  const Splat3 *pnt2splat3,
  const float *transform_world2ndc,
  const uint32_t img_w,
  const uint32_t img_h)
{
    int i_pnt = blockDim.x * blockIdx.x + threadIdx.x;
    if( i_pnt >= num_pnt ){ return; }
    //
    const auto pos_world = pnt2splat3[i_pnt].xyz;
    const cuda::std::array<float,9> world2ndc = mat4_col_major::jacobian_transform(transform_world2ndc, pos_world);
    const cuda::std::array<float,6> ndc2pix = mat2x3_col_major::transform_ndc2pix(img_w, img_h);
    const cuda::std::array<float,6> world2pix = mat2x3_col_major::mult_mat3_col_major(ndc2pix.data(), world2ndc.data());
    const auto pos_ndc = mat4_col_major::transform_homogeneous(
        transform_world2ndc, pos_world);
    const float pos_scrn[3] = {pos_ndc[0], pos_ndc[1], 1.f};
    const auto pos_pix = mat2x3_col_major::mult_vec3(ndc2pix.data(), pos_scrn);
    const cuda::std::array<float,3> sig = mat2_sym::projected_spd_mat3(
        world2pix.data(),
        pnt2splat3[i_pnt].quaternion,
        pnt2splat3[i_pnt].scale);
    const cuda::std::array<float,3> sig_inv = mat2_sym::safe_inverse_preserve_positive_definiteness(sig.data(), 1.0e-5f);
    const cuda::std::array<float,4> _aabb0 = mat2_sym::aabb2(sig_inv.data());
    const cuda::std::array<float,4> _aabb1 = aabb2::scale(_aabb0.data(), 3.f);
    const cuda::std::array<float,4> aabb = aabb2::translate(_aabb1.data(), pos_pix.data());
    //
    pnt2splat2[i_pnt].ndc_z = pos_ndc[2];
    pnt2splat2[i_pnt].pos_pix[0] = pos_pix[0];
    pnt2splat2[i_pnt].pos_pix[1] = pos_pix[1];
    pnt2splat2[i_pnt].sig_inv[0] = sig_inv[0];
    pnt2splat2[i_pnt].sig_inv[1] = sig_inv[1];
    pnt2splat2[i_pnt].sig_inv[2] = sig_inv[2];
    pnt2splat2[i_pnt].aabb[0] = aabb[0];
    pnt2splat2[i_pnt].aabb[1] = aabb[1];
    pnt2splat2[i_pnt].aabb[2] = aabb[2];
    pnt2splat2[i_pnt].aabb[3] = aabb[3];
    pnt2splat2[i_pnt].alpha = pnt2splat3[i_pnt].opacity;
    pnt2splat2[i_pnt].rgb[0] = pnt2splat3[i_pnt].rgb_dc[0];
    pnt2splat2[i_pnt].rgb[1] = pnt2splat3[i_pnt].rgb_dc[1];
    pnt2splat2[i_pnt].rgb[2] = pnt2splat3[i_pnt].rgb_dc[2];
}


__global__
void rasterize_splat_using_tile(
    uint32_t img_w,
    uint32_t img_h,
    float* d_pix2rgb,
    uint32_t tile_w,
    uint32_t tile_h,
    uint32_t tile_size,
    const uint32_t* d_tile2idx,
    const uint32_t* d_idx2pnt,
    const Splat2* d_pnt2splat)
{
    const uint32_t ix = blockDim.x * blockIdx.x + threadIdx.x;
    if( ix >= img_w ){ return; }
    //
    const uint32_t iy = blockDim.y * blockIdx.y + threadIdx.y;
    if( iy >= img_h ){ return; }
    // const uint32_t i_pix = iy * img_w + ix;
    //
    const uint32_t i_tile = (iy / tile_size) * tile_w + (ix / tile_size);
    const float t[2] = {float(ix) + 0.5f, float(iy) + 0.5f};
    float alpha_sum = 0.f;
    float alpha_occu = 1.f;
    // iterate front (z large) to back (z small)
    const uint32_t num_pnt = d_tile2idx[i_tile+1] - d_tile2idx[i_tile];
    for (uint32_t iidx=0;iidx<num_pnt;++iidx) {
        uint32_t idx = d_tile2idx[i_tile] + num_pnt - 1 - iidx;
        const uint32_t i_pnt = d_idx2pnt[idx];
        const Splat2& pnt2 = d_pnt2splat[i_pnt];
        // front to back
        if( !aabb2::is_inlcude_point(pnt2.aabb, t) ){
            continue;
        }
        const float t0[2] = {t[0] - pnt2.pos_pix[0], t[1] - pnt2.pos_pix[1]};
        float _e = mat2_sym::mult_vec_from_both_sides(pnt2.sig_inv, t0, t0);
        float e = expf(-0.5 * _e) * pnt2.alpha;
        float e_out = alpha_occu * e;
        d_pix2rgb[(iy * img_w + ix) * 3 + 0] += pnt2.rgb[0] * e_out;
        d_pix2rgb[(iy * img_w + ix) * 3 + 1] += pnt2.rgb[1] * e_out;
        d_pix2rgb[(iy * img_w + ix) * 3 + 2] += pnt2.rgb[2] * e_out;
        alpha_occu *= 1.f - e;
        alpha_sum += e_out;
        if( alpha_sum > 0.999 ){
            break;
        }
    }
}

__global__
void count_splat_in_tile(
  uint32_t num_pnt,
  const Splat2* pnt2splat,
  uint32_t* tile2ind,
  uint32_t* pnt2ind,
  uint32_t tile_w,
  uint32_t tile_h,
  uint32_t tile_size)
{
    int i_pnt = blockDim.x * blockIdx.x + threadIdx.x;
    if( i_pnt >= num_pnt ){ return; }
    //
    const Splat2& splat = pnt2splat[i_pnt];
    const float* aabb = splat.aabb;
    const cuda::std::array<float,4> aabb0 {aabb[0], aabb[1], aabb[2], aabb[3]};
    //
    tile_acceleration::count_splat_in_tile(
        i_pnt, aabb0,
        tile2ind, pnt2ind,
        tile_w, tile_h, tile_size);
}

__global__
void fill_index_info(
  uint32_t num_pnt,
  const Splat2* pnt2splat,
  const uint32_t* pnt2idx,
  uint64_t* idx2tiledepth,
  uint32_t* idx2pnt,
  uint32_t tile_w,
  uint32_t tile_h,
  uint32_t tile_size)
{
    int i_pnt = blockDim.x * blockIdx.x + threadIdx.x;
    if( i_pnt >= num_pnt ){ return; }
    //
    const Splat2& splat = pnt2splat[i_pnt];
    const float* aabb = splat.aabb;
    const cuda::std::array<float,4> aabb0 {aabb[0], aabb[1], aabb[2], aabb[3]};
    tile_acceleration::fill_index_info(
        i_pnt, aabb0, splat.ndc_z,
        pnt2idx, idx2tiledepth, idx2pnt,
        tile_w, tile_h, tile_size);
}



}